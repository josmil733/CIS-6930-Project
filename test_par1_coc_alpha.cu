#include "hip/hip_runtime.h"
// Circle of Confusion Diameter and Alpha Weight Calculations for Scattering Algorithm

// This is the first optimization of the parallel algorithm.

/* TO RUN THIS PROGRAM:
	1) nvcc par1_coc_alpha.cu -o par1_coc_alpha
	2) ./par1_coc_alpha <input_file_name.ext> <output_file_name.ext>
	3) If you want to record the execution time of the program over several repetitions (as specified in the #define REPS directive), simply enter any string as the third command line argument. If you don't give a third command line argument, the program will not record its execution time.

*/


#include <stdio.h>
#include <math.h>
#include <iostream>
#include <time.h>
#include "cudaCheck.cuh"

#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

#define WFF 36 // from Liu et. al, Equation 18
#define WSS 60 //no reference frame for this number, this is a random guess
//#define RAND_MAX 255
#define REPS 20

const int TILE_WIDTH = 16;



__global__ void getDepth(char *input_name, float depth[], const unsigned char* depthData, int *xPtr, int *yPtr){

	int x = threadIdx.x + blockDim.x*blockIdx.x;
	int y = threadIdx.y + blockDim.y*blockIdx.y;
	float meters_cnvt = 0.588235; //8-bit representation to meters conversion 					// = 150/255
					// put in shared memory for optimization
	depth[ x*(*xPtr)+y ] = depthData[4*(x*(*xPtr)+y)]* meters_cnvt;//not coalesced; would need to switch roles of x and y, or something similar.

}



__global__ void coc_alpha(float* caPtr, char *output_name, int *xPtr, int *yPtr, float* depthMap, int ncols, float ap_diam, float f, float d_focus){

	float f_f = f*WFF/WSS;//can write this one and next line into shared memory as an optimization
	float F = f/ap_diam;

	int x = threadIdx.x + blockDim.x*blockIdx.x;
	int y = threadIdx.y + blockDim.y*blockIdx.y;

	*(caPtr+2*(x*(*xPtr)+y) ) = abs(ncols*pow(WSS*f_f,2)*(d_focus-*(depthMap + x*(*xPtr)+y))/(WFF*F*(*(depthMap + x*(*xPtr)+y))*(d_focus*WFF - f_f*WSS)));
	*(caPtr+2*(x*(*xPtr)+y)+1) = 4/pow(*(caPtr+2*(x*(*xPtr)+y) ), 2);

}



/*__global__ void time(char *input_file, char *output_file, float* depthPtr, int* xPtr, int* yPtr){
	clock_t start[REPS], end[REPS];
	//clock_t start, end;
	clock_t total = 0; //counts total time
	for(int i=0; i < REPS; i++){
		start[i] = clock();

		depthPtr = getDepth(input_file, depthPtr, xPtr, yPtr);

		float* caPtr = (float *)malloc(sizeof(float)*2*(*xPtr)*(*yPtr));
		caPtr = coc_alpha(caPtr, output_file, xPtr, yPtr, depthPtr, 3, 2.5, 5.5, 4.5);
		end[i] = clock() - start[i];
		total += end[i];
	}

	printf("The program took an average of %.4f seconds per repetition, averaged across %d repetitions.\n", (float)total/(CLOCKS_PER_SEC*REPS), REPS);

}

*/

int main(int argc, char *argv[]){

	int *h_xPtr; int* h_yPtr;
	h_xPtr = (int *) malloc(sizeof(int));
	h_yPtr = (int *) malloc(sizeof(int));
	int n;

	unsigned char* h_depthData = stbi_load(argv[1], h_xPtr, h_yPtr, &n, 0);//must run on host first so that size of d_depthPtr can be determined before allocation
	
	int n_pixels = (*h_xPtr)*(*h_yPtr);
	unsigned char* h_depthPtr = (unsigned char*) malloc(sizeof(unsigned char)*n_pixels);
	
	dim3 dimGrid(ceil((float)(*h_xPtr)/(float)TILE_WIDTH), ceil((float)(*h_yPtr)/(float)TILE_WIDTH), 1);
	dim3 dimBlock(TILE_WIDTH, TILE_WIDTH, 1);
	float *d_depthPtr;
	unsigned char* d_depthData;
	int *d_xPtr, *d_yPtr;
	
	cudaCheck( hipMalloc((void**)&d_depthPtr, sizeof(float)*n_pixels) );
	cudaCheck( hipMalloc((void**)&d_depthData, 4*sizeof(unsigned char)*n_pixels) );
	cudaCheck( hipMalloc((void**)&d_xPtr, sizeof(int)) );
	cudaCheck( hipMalloc((void**)&d_yPtr, sizeof(int)) );

	hipMemcpy(d_xPtr, h_xPtr, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_yPtr, h_yPtr, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_depthData, h_depthData, 4*sizeof(unsigned char)*n_pixels, hipMemcpyHostToDevice);

	getDepth<<<dimGrid,dimBlock>>>(argv[1], d_depthPtr, d_depthData, d_xPtr, d_yPtr);
	hipError_t err2 = hipGetLastError();
	if(err2 != hipSuccess) printf("Error: %s\n", hipGetErrorString(err2));
	
	hipDeviceSynchronize();
	hipMemcpy(h_depthPtr, d_depthPtr, sizeof(float)*n_pixels, hipMemcpyDeviceToHost);
	cudaCheck( hipMemcpy(h_depthPtr, d_depthPtr, sizeof(unsigned char)*n_pixels, hipMemcpyDeviceToHost) );
	hipError_t err3 = hipGetLastError();
	if(err3 != hipSuccess){
		 printf("Error: %s\n", hipGetErrorString(err2));
		hipFree(d_xPtr);
		hipFree(d_yPtr);
		hipFree(d_depthPtr);
		hipFree(d_depthData);
	}

	float* h_caPtr = (float *)malloc(sizeof(float)*2*n_pixels);
	float* d_caPtr;
	hipMalloc(&d_caPtr, sizeof(float)*2*n_pixels);
	coc_alpha<<<dimGrid,dimBlock>>>(d_caPtr, argv[2], d_xPtr, d_yPtr, d_depthPtr, 3, 2.5, 5.5, 4.5);
	hipDeviceSynchronize();	
	cudaCheck( hipMemcpy(h_caPtr, d_caPtr, sizeof(float)*2*n_pixels, hipMemcpyDeviceToHost) );

	hipFree(d_xPtr);
	hipFree(d_yPtr);
	hipFree(d_depthPtr);
	hipFree(d_depthData);
	hipFree(d_caPtr);

	return 1;
}
